#include <stdio.h>
#include <stdlib.h>
#include <assert.h>
#include <getopt.h>
#include <hip/hip_runtime.h>
#include <hipsolver.h>

#include "common.h"
#include "lud_kernel"

#ifdef RD_WG_SIZE_0_0
    #define BLOCK_SIZE RD_WG_SIZE_0_0
#elif defined(RD_WG_SIZE_0)
    #define BLOCK_SIZE RD_WG_SIZE_0
#elif defined(RD_WG_SIZE)
    #define BLOCK_SIZE RD_WG_SIZE
#else
    #define BLOCK_SIZE 16
#endif

static int do_verify = 0, use_rodina = 0;

static struct option long_options[] = {
  /* name, has_arg, flag, val */
  {"input", 1, NULL, 'i'},
  {"size", 1, NULL, 's'},
  {"verify", 0, NULL, 'v'},
  {"rodina", 0, NULL, 'r'},
  {0,0,0,0}
};

#define CHECK_CUDA(call)                                          \
  if ((call) != hipSuccess)                                      \
  {                                                               \
    fprintf(stderr, "CUDA error at %s %d\n", __FILE__, __LINE__); \
    return EXIT_FAILURE;                                          \
  }

#define CHECK_CUSOLVER(call)                                          \
  if ((call) != HIPSOLVER_STATUS_SUCCESS)                              \
  {                                                                   \
    fprintf(stderr, "cuSOLVER error at %s %d\n", __FILE__, __LINE__); \
    return EXIT_FAILURE;                                              \
  }

int main(int argc, char *argv[])
{
  printf("WG size of kernel = %d X %d\n", BLOCK_SIZE, BLOCK_SIZE);
  hipsolverHandle_t handle;
  int *devIpiv, *devInfo;
  int matrix_dim = 32; // example matrix size
  int Lwork = 0;
  int opt, option_index = 0;
  func_ret_t ret;
  const char *input_file = NULL;
  double *m, *d_m, *mm;
  stopwatch sw;

  while ((opt = getopt_long(argc, argv, "::vs:i:",
                            long_options, &option_index)) != -1)
  {
    switch (opt)
    {
    case 'i':
      input_file = optarg;
      break;
    case 'v':
      do_verify = 1;
      break;
    case 'r':
      use_rodina = 1;
      break;
    case 's':
      matrix_dim = atoi(optarg);
      printf("Generate input matrix internally, size =%d\n", matrix_dim);
      // fprintf(stderr, "Currently not supported, use -i instead\n");
      // fprintf(stderr, "Usage: %s [-v] [-s matrix_size|-i input_file]\n", argv[0]);
      // exit(EXIT_FAILURE);
      break;
    case '?':
      fprintf(stderr, "invalid option\n");
      break;
    case ':':
      fprintf(stderr, "missing argument\n");
      break;
    default:
      fprintf(stderr, "Usage: %s [-v] [-s matrix_size|-i input_file]\n",
              argv[0]);
      exit(EXIT_FAILURE);
    }
  }

  if ((optind < argc) || (optind == 1))
  {
    fprintf(stderr, "Usage: %s [-v] [-s matrix_size|-i input_file]\n", argv[0]);
    exit(EXIT_FAILURE);
  }

  if (input_file)
  {
    printf("Reading matrix from file %s\n", input_file);
    ret = create_matrix_from_file(&m, input_file, &matrix_dim);
    if (ret != RET_SUCCESS)
    {
      m = NULL;
      fprintf(stderr, "error create matrix from file %s\n", input_file);
      exit(EXIT_FAILURE);
    }
  }
  else if (matrix_dim)
  {
    printf("Creating matrix internally size=%d\n", matrix_dim);
    ret = create_matrix(&m, matrix_dim);
    if (ret != RET_SUCCESS)
    {
      m = NULL;
      fprintf(stderr, "error create matrix internally size=%d\n", matrix_dim);
      exit(EXIT_FAILURE);
    }
  }

  else
  {
    printf("No input file specified!\n");
    exit(EXIT_FAILURE);
  }
  if (do_verify)
  {
    printf("Before LUD\n");
    // print_matrix(m, matrix_dim);
    matrix_duplicate(m, &mm, matrix_dim);
  }

  // Allocate the device matrix
  CHECK_CUDA(hipMalloc((void **)&d_m, matrix_dim * matrix_dim * sizeof(double)));
  printf("Performing LU decomposition\n");
  stopwatch_start(&sw);

  // Copy the host matrix to the device
  CHECK_CUDA(hipMemcpy(d_m, m, matrix_dim * matrix_dim * sizeof(double), hipMemcpyHostToDevice));

  if(use_rodina){
    lud_cuda(d_m, matrix_dim);
  }
  else {
    // Create the cuSOLVER handle
    CHECK_CUSOLVER(hipsolverDnCreate(&handle));

    // Allocate the pivot array and info parameter on the device
    CHECK_CUDA(hipMalloc((void **)&devIpiv, matrix_dim * sizeof(int)));
    CHECK_CUDA(hipMalloc((void **)&devInfo, sizeof(int)));

    // Compute the LU decomposition
    CHECK_CUSOLVER(hipsolverDnDgetrf_bufferSize(handle, matrix_dim, matrix_dim, d_m, matrix_dim, &Lwork));
    double *devWork = NULL;
    CHECK_CUDA(hipMalloc((void **)&devWork, sizeof(double) * Lwork));
    CHECK_CUSOLVER(hipsolverDnDgetrf(handle, matrix_dim, matrix_dim, d_m, matrix_dim, devWork, devIpiv, devInfo));
  }

  // Copy the result back to the host
  CHECK_CUDA(hipMemcpy(m, d_m, matrix_dim * matrix_dim * sizeof(double), hipMemcpyDeviceToHost));


  stopwatch_stop(&sw);
  printf("Time consumed(ms): %lf\n", 1000*get_interval_by_sec(&sw));

  printf("LU decomposition completed\n");

  if(use_rodina == 0) {}
    int hostInfo;
    CHECK_CUDA(hipMemcpy(&hostInfo, devInfo, sizeof(int), hipMemcpyDeviceToHost));
    printf("Devinfo: %d\n", hostInfo);
  }


  if (do_verify){
    printf("After LUD\n");
    //print_matrix(m, matrix_dim);
    //print_matrix(mm, matrix_dim);
    printf(">>>Verify<<<<\n");
    lud_verify(mm, m, matrix_dim, use_rodina);
    free(mm);
  }

  // Cleanup
  CHECK_CUDA(hipFree(d_m));
  if(use_rodina == 0) {
    CHECK_CUDA(hipFree(devIpiv));
    CHECK_CUDA(hipFree(devInfo));
    CHECK_CUDA(hipFree(devWork));
    hipsolverDnDestroy(handle);
  }
  free(m);

  return 0;
}